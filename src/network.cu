#include "hip/hip_runtime.h"
// network.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "network.h"
#include "utils.h"
#include <omp.h> // Ensure this is included for OpenMP

REAL Sunspots[NUM_YEARS];
REAL Sunspots_[NUM_YEARS];
REAL Mean;
REAL TrainError;
REAL TrainErrorPredictingMean;
REAL TestError;
REAL TestErrorPredictingMean;

const int Units[] = { };

// initialization
void GenerateNetwork(NET* Net) {
    INT l, i;

    Net->Layer = (LAYER**) calloc(NUM_LAYERS, sizeof(LAYER*));

    for (l = 0; l < NUM_LAYERS; l++) {
        Net->Layer[l] = (LAYER*) malloc(sizeof(LAYER));

        Net->Layer[l]->Units = Units[l];
        Net->Layer[l]->Output = (REAL*) calloc(Units[l] + 1, sizeof(REAL));
        Net->Layer[l]->Error = (REAL*) calloc(Units[l] + 1, sizeof(REAL));
        Net->Layer[l]->Weight = (REAL**) calloc(Units[l] + 1, sizeof(REAL*));
        Net->Layer[l]->WeightSave = (REAL**) calloc(Units[l] + 1, sizeof(REAL*));
        Net->Layer[l]->dWeight = (REAL**) calloc(Units[l] + 1, sizeof(REAL*));
        Net->Layer[l]->Output[0] = BIAS;

        if (l != 0) {
            for (i = 1; i <= Units[l]; i++) {
                Net->Layer[l]->Weight[i] = (REAL*) calloc(Units[l - 1] + 1, sizeof(REAL));
                Net->Layer[l]->WeightSave[i] = (REAL*) calloc(Units[l - 1] + 1, sizeof(REAL));
                Net->Layer[l]->dWeight[i] = (REAL*) calloc(Units[l - 1] + 1, sizeof(REAL));
            }
        }
    }
    Net->InputLayer = Net->Layer[0];
    Net->OutputLayer = Net->Layer[NUM_LAYERS - 1];
    Net->Alpha = 0.9;
    Net->Eta = 0.25;
    Net->Gain = 1;
}

// initialize weights randomly
void RandomWeights(NET* Net) {
    INT l, i, j;

    for (l = 1; l < NUM_LAYERS; l++) {
        for (i = 1; i <= Net->Layer[l]->Units; i++) {
            for (j = 0; j <= Net->Layer[l - 1]->Units; j++) {
                Net->Layer[l]->Weight[i][j] = RandomEqualREAL(-0.5, 0.5);
            }
        }
    }
}

// set input values
void SetInput(NET* Net, REAL* Input) {
    INT i;

    for (i = 1; i <= Net->InputLayer->Units; i++) {
        Net->InputLayer->Output[i] = Input[i - 1];
    }
}

// get output values
void GetOutput(NET* Net, REAL* Output) {
    INT i;

    for (i = 1; i <= Net->OutputLayer->Units; i++) {
        Output[i - 1] = Net->OutputLayer->Output[i];
    }
}


// propagate signals through a layer
void PropagateLayer(NET* Net, LAYER* Lower, LAYER* Upper) {
    INT i, j;
    REAL Sum;

    for (i = 1; i <= Upper->Units; i++) {
        Sum = 0;
        for (j = 0; j <= Lower->Units; j++) {
            Sum += Upper->Weight[i][j] * Lower->Output[j];
        }
        Upper->Output[i] = 1 / (1 + exp(-Net->Gain * Sum));
    }
}

// propagate signals through the network
void PropagateNet(NET* Net) {
    INT l;

    for (l = 0; l < NUM_LAYERS - 1; l++) {
        PropagateLayerCUDA(Net, Net->Layer[l], Net->Layer[l + 1]);
    }
}

// compute output error
void ComputeOutputError(NET* Net, REAL* Target) {
    INT i;
    REAL Out, Err;

    Net->Error = 0;
    for (i = 1; i <= Net->OutputLayer->Units; i++) {
        Out = Net->OutputLayer->Output[i];
        Err = Target[i - 1] - Out;
        Net->OutputLayer->Error[i] = Net->Gain * Out * (1 - Out) * Err;
        Net->Error += 0.5 * sqr(Err);
    }
}

// backpropagate error through a layer
void BackpropagateLayer(NET* Net, LAYER* Upper, LAYER* Lower) {
    INT i, j;
    REAL Out, Err;

    for (i = 1; i <= Lower->Units; i++) {
        Out = Lower->Output[i];
        Err = 0;
        for (j = 1; j <= Upper->Units; j++) {
            Err += Upper->Weight[j][i] * Upper->Error[j];
        }
        Lower->Error[i] = Net->Gain * Out * (1 - Out) * Err;
    }
}

// backpropagate error through the network
void BackpropagateNet(NET* Net) {
    INT l;

    for (l = NUM_LAYERS - 1; l > 1; l--) {
        BackpropagateLayerCUDA(Net, Net->Layer[l], Net->Layer[l - 1]);
    }
}

// adjust weights based on errors
void AdjustWeights(NET* Net) {
    INT l, i, j;
    REAL Out, Err, dWeight;

    for (l = 1; l < NUM_LAYERS; l++) {
        for (i = 1; i <= Net->Layer[l]->Units; i++) {
            for (j = 0; j <= Net->Layer[l - 1]->Units; j++) {
                Out = Net->Layer[l - 1]->Output[j];
                Err = Net->Layer[l]->Error[i];
                dWeight = Net->Layer[l]->dWeight[i][j];
                Net->Layer[l]->Weight[i][j] += Net->Eta * Err * Out + Net->Alpha * dWeight;
                Net->Layer[l]->dWeight[i][j] = Net->Eta * Err * Out;
            }
        }
    }
}

// simulate the network
void SimulateNet(NET* Net, REAL* Input, REAL* Output, REAL* Target, BOOL Training) {
    SetInput(Net, Input);
    PropagateNet(Net);
    GetOutput(Net, Output);

    ComputeOutputError(Net, Target);
    if (Training) {
        BackpropagateNet(Net);
        AdjustWeights(Net);
    }
}

// train the network
void TrainNet(NET* Net, INT Epochs) {
    INT Year, n;
    REAL Output[M];

    #pragma omp parallel for private(Year, Output) schedule(dynamic)
    for (n = 0; n < Epochs * TRAIN_YEARS; n++) {
        Year = RandomEqualINT(TRAIN_LWB, TRAIN_UPB);
        SimulateNet(Net, &(Sunspots[Year - N]), Output, &(Sunspots[Year]), TRUE);
    }
}

// test the network
void TestNet(NET* Net) {
    INT Year;
    REAL Output[M];

    TrainError = 0;
    for (Year = TRAIN_LWB; Year <= TRAIN_UPB; Year++) {
        SimulateNet(Net, &(Sunspots[Year - N]), Output, &(Sunspots[Year]), FALSE);
        TrainError += Net->Error;
    }
    TestError = 0;
    for (Year = TEST_LWB; Year <= TEST_UPB; Year++) {
        SimulateNet(Net, &(Sunspots[Year - N]), Output, &(Sunspots[Year]), FALSE);
        TestError += Net->Error;
    }
    fprintf(f, "\nNMSE is %0.3f on Training Set and %0.3f on Test Set",
        TrainError / TrainErrorPredictingMean,
        TestError / TestErrorPredictingMean);
}

// evaluate the network
void EvaluateNet(NET* Net) {
    INT Year;
    REAL Output[M];
    REAL Output_[M];

    fprintf(f, "\n\n\n");
    fprintf(f, "Year    Sunspots    Open-Loop Prediction    Closed-Loop Prediction\n");
    fprintf(f, "\n");
    for (Year = EVAL_LWB; Year <= EVAL_UPB; Year++) {
        SimulateNet(Net, &(Sunspots[Year - N]), Output, &(Sunspots[Year]), FALSE);
        SimulateNet(Net, &(Sunspots_[Year - N]), Output_, &(Sunspots_[Year]), FALSE);
        Sunspots_[Year] = Output_[0];
        fprintf(f, "%d       %0.3f                   %0.3f                     %0.3f\n",
            FIRST_YEAR + Year,
            Sunspots[Year],
            Output[0],
            Output_[0]);
    }
}
