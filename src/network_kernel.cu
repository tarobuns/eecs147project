#include "hip/hip_runtime.h"
// network_kernel.cu
#include "network.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256

#define cudaCheckError() {                                          \
    hipError_t e=hipGetLastError();                                \
    if(e!=hipSuccess) {                                             \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                                          \
    }                                                                \
}

// the kernel for forward propagation
__global__ void PropagateLayerKernel(REAL* Weight, REAL* Input, REAL* Output, int InputSize, int OutputSize, REAL Gain) {
    __shared__ REAL SharedInput[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < InputSize) {
        SharedInput[tid] = Input[tid];
    }
    __syncthreads();

    if (idx < OutputSize) {
        REAL Sum = 0.0;
        for (int j = 0; j < InputSize; j++) {
            Sum += Weight[idx * InputSize + j] * SharedInput[j];
        }
        Output[idx] = 1.0 / (1.0 + exp(-Gain * Sum));
    }
}

// Kernel for backpropagation
__global__ void BackpropagateLayerKernel(REAL* Weight, REAL* Output, REAL* Error, REAL* NextError, int InputSize, int OutputSize, REAL Gain) {
    __shared__ REAL SharedOutput[BLOCK_SIZE];
    __shared__ REAL SharedNextError[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < OutputSize) {
        SharedOutput[tid] = Output[tid];
        SharedNextError[tid] = NextError[tid];
    }
    __syncthreads();

    if (idx < InputSize) {
        REAL Err = 0.0;
        for (int j = 0; j < OutputSize; j++) {
            Err += Weight[j * InputSize + idx] * SharedNextError[j];
        }
        Error[idx] = Gain * SharedOutput[idx] * (1.0 - SharedOutput[idx]) * Err;
    }
}

// using CUDA to propogate through a layer
void PropagateLayerCUDA(NET* Net, LAYER* Lower, LAYER* Upper) {
    int InputSize = Lower->Units + 1; // +1 for the bias
    int OutputSize = Upper->Units;

    // allocate memory on the device
    REAL *d_Weight, *d_Input, *d_Output;
    hipMalloc(&d_Weight, OutputSize * InputSize * sizeof(REAL)); cudaCheckError();
    hipMalloc(&d_Input, InputSize * sizeof(REAL)); cudaCheckError();
    hipMalloc(&d_Output, OutputSize * sizeof(REAL)); cudaCheckError();

    // opy data to the device
    hipMemcpy(d_Weight, Upper->Weight[1], OutputSize * InputSize * sizeof(REAL), hipMemcpyHostToDevice); cudaCheckError();
    hipMemcpy(d_Input, Lower->Output, InputSize * sizeof(REAL), hipMemcpyHostToDevice); cudaCheckError();

    // define the number of threads and blocks
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (OutputSize + threadsPerBlock - 1) / threadsPerBlock;

    // launch the kernel
    PropagateLayerKernel<<<blocksPerGrid, threadsPerBlock>>>(d_Weight, d_Input, d_Output, InputSize, OutputSize, Net->Gain); cudaCheckError();

    // synchronize the device
    hipDeviceSynchronize(); cudaCheckError();

    // copy the result back to the host
    hipMemcpy(Upper->Output + 1, d_Output, OutputSize * sizeof(REAL), hipMemcpyDeviceToHost); cudaCheckError();

    // free device memory
    hipFree(d_Weight); cudaCheckError();
    hipFree(d_Input); cudaCheckError();
    hipFree(d_Output); cudaCheckError();
}

// function to backpropagate through a layer using CUDA
void BackpropagateLayerCUDA(NET* Net, LAYER* Upper, LAYER* Lower) {
    int InputSize = Lower->Units + 1; // +1 for the bias
    int OutputSize = Upper->Units;

    // allocate memory on the device
    REAL *d_Weight, *d_Output, *d_Error, *d_NextError;
    hipMalloc(&d_Weight, OutputSize * InputSize * sizeof(REAL)); cudaCheckError();
    hipMalloc(&d_Output, InputSize * sizeof(REAL)); cudaCheckError();
    hipMalloc(&d_Error, InputSize * sizeof(REAL)); cudaCheckError();
    hipMalloc(&d_NextError, OutputSize * sizeof(REAL)); cudaCheckError();

    // copy data to the device
    hipMemcpy(d_Weight, Upper->Weight[1], OutputSize * InputSize * sizeof(REAL), hipMemcpyHostToDevice); cudaCheckError();
    hipMemcpy(d_Output, Lower->Output, InputSize * sizeof(REAL), hipMemcpyHostToDevice); cudaCheckError();
    hipMemcpy(d_NextError, Upper->Error + 1, OutputSize * sizeof(REAL), hipMemcpyHostToDevice); cudaCheckError();

    // define the number of threads and blocks
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (InputSize + threadsPerBlock - 1) / threadsPerBlock;

    // launch the kernel
    BackpropagateLayerKernel<<<blocksPerGrid, threadsPerBlock>>>(d_Weight, d_Output, d_Error, d_NextError, InputSize, OutputSize, Net->Gain); cudaCheckError();

    // synchronize the device
    hipDeviceSynchronize(); cudaCheckError();

    // copy the result back to the host
    hipMemcpy(Lower->Error + 1, d_Error, InputSize * sizeof(REAL), hipMemcpyDeviceToHost); cudaCheckError();

    // free device memory
    hipFree(d_Weight); cudaCheckError();
    hipFree(d_Output); cudaCheckError();
    hipFree(d_Error); cudaCheckError();
    hipFree(d_NextError); cudaCheckError();
}
